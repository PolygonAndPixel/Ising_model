#include "hip/hip_runtime.h"
#include <time.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "include/hpc_helpers.hpp"
#include "include/binary_IO.hpp"
#include "include/bitmap_IO.hpp"
#include <iostream>
#include <fstream>

#define LENGTH 1<<8 // 4096, 1<<15 works too, 1024 gives weird results
#define BLOCKSQRT 11 // 
#define BLOCKSIZE (BLOCKSQRT*BLOCKSQRT)
// Given n threads
// Each spin needs 4 neighbours -> padding of 4*2*sqrt(n) for each side
// e.g. n = 1024 threads
// Checkerboard update in quadratic manner -> sqrt(n) updates on each side
// Twice because of checkerboard and times 4 for each side
// 256 padded values + n*2 (checkerboard) for actual updates
// 2304 values e.g. floats
// 2304 * 32 bits / (8 * 1024) = 9 KB (fits nicely in shared memory)
// TODO: Create a register variant with warp shuffle after each update
// Register size: 256 KB / n = 64 floats for each thread
// Each thread needs in checkerboard only 32 values where we divide
// the grid within each block in tiles in ch

#define REGISTER_SIZE 32
#define THREAD_TILE_WIDTH 4 // Register size is 4*4 + padding (16 values)
#define N_TEMPS 26 // The number of increases in temperature // default 26
#define DELTA_T 0.1 // The amount of Kelvin to increase for every N_TEMPS // default 0.1
#define RUNS 5 // Reduce the lattice size by half // 7
#define ITERATIONS 4 // Iterations within a window before moving on 100 // default 4
#define OVERALL_ITERATIONS 128 // ((LENGTH * 2 + THREAD_TILE_WIDTH - 1) / THREAD_TILE_WIDTH) // Iterations of all blocks // default 128, is multiplied with length*length/2 later
#define SLIDING_ITERATIONS 32 // ((LENGTH * 2 + THREAD_TILE_WIDTH + 15) / (THREAD_TILE_WIDTH + 16)) // Sliding window within a block (left to right) for one circle // default 8
// the stride of the windows is THREAD_TILE_WIDTH/2
#define RUNS_AVERAGE 1 // Runs to calculate the average (just to be sure) // default 1
#define DATA_PER_RUN 100 // Number of times where the current state should be flushed to disk // default 100, max 131072
#define RUNS_BEFORE_FLUSH 100 // Number of iterations between two flushes // default 100
#define DUMP_IMG 0 // Set to 0 if you don't want any images
#define MC 0 // Use this for data on xmgrace

__device__ void shuffle_spins(float * register_spins, bool black)
{
    // Shuffle up right hand side
    #pragma unroll
    for(int i=0; i < THREAD_TILE_WIDTH; i+=2)
    {
        int to_shuffle = THREAD_TILE_WIDTH*2 + (i+black)*(THREAD_TILE_WIDTH+2);
        float new_pad = __shfl_up(register_spins[to_shuffle], 1);
        int shuffle_here = THREAD_TILE_WIDTH + (i+black)*(THREAD_TILE_WIDTH+2);
        if(threadIdx.x%32 != 0)
            register_spins[shuffle_here] = new_pad;
    }
    // Shuffle down left hand side
    bool black2 = !black;
    #pragma unroll
    for(int i=0; i < THREAD_TILE_WIDTH; i+=2)
    {
        int to_shuffle = THREAD_TILE_WIDTH + 1 
            + (i+black2)*(THREAD_TILE_WIDTH+2);
        float new_pad = __shfl_down(register_spins[to_shuffle], 1);
        int shuffle_here = THREAD_TILE_WIDTH 
            + (i+black2)*(THREAD_TILE_WIDTH+2);
        if(threadIdx.x%32 != 31)
            register_spins[shuffle_here] = new_pad;
    }
}

__device__ float generate(hiprandState* globalState, int ind)
{
    //copy state to local mem
    hiprandState localState = globalState[ind];
    //apply uniform distribution with calculated random
    float rndval = hiprand_uniform( &localState );
    //update state
    globalState[ind] = localState;
    //return value
    return rndval;
}

__global__ void initialise_curand_on_kernels(hiprandState * state, 
    unsigned long seed)
{
    int idx = threadIdx.x + threadIdx.y * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ void update_spins(float * register_spins, bool black, 
    hiprandState* globalState, float temperature)
{   
    int idx = threadIdx.x + threadIdx.y * blockDim.x;
    // Update first row
    #pragma unroll
    for(int col=0; col<THREAD_TILE_WIDTH/2; col++)
    {
        int i = THREAD_TILE_WIDTH+1 + !black + 2*col;
        // Check boundaries
        // up
        float energy = register_spins[i-THREAD_TILE_WIDTH-1];
        // down
        energy += register_spins[i+THREAD_TILE_WIDTH+2];
        // left
        energy += register_spins[i-1];
        // right
        energy += register_spins[i+1];
        energy *= register_spins[i];
        if(energy < 0)
        {
            register_spins[i] *= (-1);
        } else 
        {
            float p = generate(globalState, idx);
            if(p < expf(-temperature*2*energy))
            {
                register_spins[i] *= (-1);
            }  
        }
    }
    // Update intermediate
    #pragma unroll
    for(int col=0; col<THREAD_TILE_WIDTH/2; col++)
    {
        int i = 2*THREAD_TILE_WIDTH+3 + 2*col + black;
        // Check boundaries
        float energy = register_spins[i-THREAD_TILE_WIDTH-2];
        energy += register_spins[i+THREAD_TILE_WIDTH+2];
        energy += register_spins[i-1];
        energy += register_spins[i+1];
        energy *= register_spins[i];
        if(energy < 0)
        {
            register_spins[i] *= (-1);
        } else 
        {
            float p = generate(globalState, idx);
            if( p < expf(-temperature*2*energy))
            {
                register_spins[i] *= (-1);
            }
        }
    }
    
    #pragma unroll
    for(int col=0; col<THREAD_TILE_WIDTH/2; col++)
    {
        int i = 2*THREAD_TILE_WIDTH+3 + THREAD_TILE_WIDTH+2 + 2*col + !black;
        // Check boundaries
        float energy = register_spins[i-THREAD_TILE_WIDTH-2];
        energy += register_spins[i+THREAD_TILE_WIDTH+2];
        energy += register_spins[i-1];
        energy += register_spins[i+1];
        energy *= register_spins[i];
        if(energy < 0)
        {
            register_spins[i] *= (-1);
        } else 
        {
            float p = generate(globalState, idx);
            if( p < expf(-temperature*2*energy))
            {
                register_spins[i] *= (-1);
            }
        }
    }
    
    // Update bottom
    #pragma unroll
    for(int col=0; col<THREAD_TILE_WIDTH/2; col++)
    {
        int i = 2*THREAD_TILE_WIDTH+3 
            + THREAD_TILE_WIDTH/2*(THREAD_TILE_WIDTH+2) + 2*col + black;
        // Check boundaries
        float energy = register_spins[i-THREAD_TILE_WIDTH-2];
        energy += register_spins[i+THREAD_TILE_WIDTH+1];
        energy += register_spins[i-1];
        energy += register_spins[i+1];
        energy *= register_spins[i];
        
        //float energy_after = energy_before * (-1);
        //energy_before *= (-1) * register_spins[i];
        // energy_after was multiplied with (-1) before, hence again.
        //energy_after *= (-1) * register_spins[i];
        if(energy < 0)
        {
            register_spins[i] *= (-1);
        } else 
        {
            float p = generate(globalState, idx);
            if( p < expf(-temperature*2*energy))
            {
                register_spins[i] *= (-1);
            }
        }
    }
}

__global__ void isis(float * spins, int length, hiprandState* globalState, 
    int sy, float temperature)
{  
    float register_spins[REGISTER_SIZE];
    // Use a sliding window 
    for(int sx=0; sx < SLIDING_ITERATIONS; sx++)
    {
        // Each block processes THREAD_TILE_WIDTH x THREAD_TILE_WIDTH as many values hence the THREAD_TILE_WIDTH
        int idx_x_global = threadIdx.x * THREAD_TILE_WIDTH 
            + THREAD_TILE_WIDTH*blockDim.x * blockIdx.x
            + THREAD_TILE_WIDTH*3/2 * sx;
        // Basically the row of the model
        int idx_y_global = THREAD_TILE_WIDTH*blockDim.y * blockIdx.y
            + THREAD_TILE_WIDTH/2 * sy;
        // Check if thread starts within the model
        if(idx_x_global < length && idx_y_global < length) 
        {
            // Load values to register. Each thread handles a tile of 4*4 values
            // with 4*4 padded values. Remember: Shuffle boundaries!
            // Upper boundary
            #pragma unroll
            for(int x=0; x < THREAD_TILE_WIDTH; x++)
            {
                int x_spin_idx = (idx_x_global + x)%length;
                int y_spin_idx = (idx_y_global - 1) * length;
                if(y_spin_idx < 0) y_spin_idx = length * (length-1); 
                register_spins[x] = spins[x_spin_idx + y_spin_idx];
            }
            // Middle part + boundaries left and right
            #pragma unroll
            for(int y=0; y < THREAD_TILE_WIDTH; y++)
            {
                for(int x=-1; x < THREAD_TILE_WIDTH+1; x++)
                {
                    int x_spin_idx = (idx_x_global + x)%length;
                    int y_spin_idx = (y + idx_y_global)%length * length;
                    int idx_register = 5 + x + y*(THREAD_TILE_WIDTH+2);
                    if(x_spin_idx < 0) x_spin_idx += length;
                    register_spins[idx_register] = 
                        spins[x_spin_idx + y_spin_idx];
                }
            }
            // Bottom boundary
            #pragma unroll
            for(int x=0; x < THREAD_TILE_WIDTH; x++)
            {
                int x_spin_idx = (idx_x_global + x)%length;
                int y_spin_idx = (idx_y_global + 
                                  THREAD_TILE_WIDTH)%length * length;
                register_spins[x] = spins[x_spin_idx + y_spin_idx];
            }
            
            bool black = 0;
            // Update the spins
            #pragma unroll
            for(int i=0; i < ITERATIONS; i++)
            {
                update_spins(register_spins, black, globalState, temperature);
                shuffle_spins(register_spins, black);
                black = !black;
            }
            
            // write back to spins
            #pragma unroll
            for(int y=0; y < THREAD_TILE_WIDTH; y++)
            {
                for(int x=0; x < THREAD_TILE_WIDTH; x++)
                {
                    int x_spin_idx = (idx_x_global + x)%length;
                    int y_spin_idx = (y + idx_y_global)%length * length;
                    int idx_register = 5 + x + y*(THREAD_TILE_WIDTH+2);
                    spins[x_spin_idx + y_spin_idx] = 
                        register_spins[idx_register];
                }
            }
        }
        __syncthreads();
    }
}

// Observables are:
// Average energy
// Average magnetization
__global__ void getObservables(float * spins, int length, float * observables)
{
    int idx_x_global = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_y_global = blockIdx.y; // The y-dimension of a block is 1.
    float my_energy = 0;
    float magnetization = 0;
    
    for(int row=idx_y_global; row<length; row+=gridDim.y)
    {
        for(int col=idx_x_global; col<length; col+=blockDim.x*gridDim.x)
        {
            magnetization += spins[col + row*length];
            /*
            my_energy -= ( spins[col + row*length] 
                         * spins[(col+1)%length + row*length]
                         + spins[col + row*length] 
                         * spins[col + ((row+1)%length)*length]
                         + spins[col + row*length] 
                         * spins[(col-1)%length + row*length]
                         + spins[col + row*length] 
                         * spins[col + ((row-1)%length)*length]);
            */
            // Simplified
            int left = (col-1)%length;
            int below = (row-1)%length;
            if(left < 0) left += length;
            if(below < 0) below += length;
            my_energy -= ( spins[(col+1)%length + row*length]
                         + spins[col + ((row+1)%length)*length]
                         + spins[left + row*length]
                         + spins[col + below*length]) 
                         * spins[col + row*length] ; 
                         
        }
    }
    atomicAdd(observables, my_energy);
    atomicAdd(observables+1, magnetization);
}

int main (int argc, char * argv[]) 
{
    hipDeviceReset();
    TIMERSTART(preparing)
    srand(time(NULL));
    //alocate space for each kernels hiprandState
    hiprandState* deviceStates;
    // TODO: Make THREAD_TILE_WIDTH = length/BLOCKSIZE. Does it work? Must not be odd!
    int length = LENGTH;
    float *all_spins = nullptr;
    if(DUMP_IMG > 0)
    {
        // Save all spins for bitmaps
        double mem = 4 * LENGTH * LENGTH;
        mem *=  DUMP_IMG;
        mem /= (1024*1024);
        printf("Extra memory: %f Mbyte\n", mem);
        //float * all_spins = new float[LENGTH*LENGTH*10];
        size_t size = sizeof(float)*LENGTH*LENGTH;
        size *= DUMP_IMG;
        
        hipHostMalloc(&all_spins, size); CUERR
    }
    
    hipMalloc(&deviceStates, sizeof(hiprandState)*BLOCKSIZE);             CUERR
    float * observables = nullptr;
    // We save later lattice size, temperature, energy, magnetization
    hipHostMalloc(&observables, sizeof(float)*4*RUNS*N_TEMPS);           CUERR
    float * stats = nullptr;
    hipHostMalloc(&stats, sizeof(float)*2*DATA_PER_RUN);                 CUERR
    float * Stats = nullptr;
    hipMalloc(&Stats, sizeof(float)*2*DATA_PER_RUN);                     CUERR
    float * Observables = nullptr;
    hipMalloc(&Observables, sizeof(float)*2);                            CUERR
    float * spins = nullptr;
    hipHostMalloc(&spins, sizeof(float)*length*length);                  CUERR
    // Generate random spins
    for(int i=0; i<length*length; i++)
    {
        int r = (rand()%2)*2 - 1;
        spins[i] = r;
    }

    float * Spins = nullptr;
    hipMalloc(&Spins, sizeof(float)*length*length);                      CUERR
    // x-dimension should be 1 or else shuffle doesn't properly work.
    int grid_x = length/(THREAD_TILE_WIDTH * BLOCKSIZE);
    int grid_y = length/THREAD_TILE_WIDTH;
    int grid_z = 1;
    if(grid_x < 1) grid_x = 1;
    dim3 gridDims(grid_x, grid_y, grid_z);
    TIMERSTOP(preparing)
    TIMERSTART(init_curand)
    //call hiprand_init on each kernel with the same random seed
    //and init the rng states
    initialise_curand_on_kernels<<<gridDims, BLOCKSIZE>>>(deviceStates,
                                                          unsigned(time(NULL))); CUERR
    TIMERSTOP(init_curand)
    TIMERSTART(H2D)
    hipMemcpy(Spins, spins, sizeof(float)*length*length, 
               hipMemcpyHostToDevice);                                   CUERR
    TIMERSTOP(H2D)
    printf("Using (%d, %d, %d) blocks and %d threads per block\n", 
        grid_x, grid_y, grid_z, BLOCKSIZE);
    printf("Iterating %d times over all blocks and %d times within a window and %d times we slide a window\n", 
        OVERALL_ITERATIONS, ITERATIONS, SLIDING_ITERATIONS);
    uint n_updates = grid_x * grid_y * grid_z 
        * BLOCKSIZE * OVERALL_ITERATIONS * ITERATIONS * SLIDING_ITERATIONS
        * THREAD_TILE_WIDTH * THREAD_TILE_WIDTH / 4; // each thread updates in a
                                                     // checkerboard way half of its registers.
    printf("Overall %u updates on a %d x %d grid\n", n_updates, length, length);
    float used_memory = sizeof(float)*length*length + sizeof(hiprandState)*BLOCKSIZE;
    used_memory /= (1024*1024);
    printf("Using %f mByte data\n", used_memory);
    
    std::ofstream fs;
    if(argc > 2)
        fs.open(argv[2]);
    else
        fs.open("cuda_log");
    if(DATA_PER_RUN > 0 && !MC) 
        fs << "lattice_size\ttemperature\tenergy\tmagnetization\n";
    
    for(int run=0; run<RUNS; run++)
    {
        length = LENGTH >> run;
        grid_x = length/(THREAD_TILE_WIDTH * BLOCKSIZE);
        grid_y = length/THREAD_TILE_WIDTH;
        if(grid_x < 1) grid_x = 1;
        dim3 gridDims(grid_x, grid_y, grid_z);
        printf("Using (%d, %d, %d) blocks and %d threads per block\n", 
        grid_x, grid_y, grid_z, BLOCKSIZE);
        for(int t=0; t<N_TEMPS; t++)
        {
            float energy = 0;
            float magnet = 0;
            float temperature = DELTA_T * t + 1; 
            if(MC) temperature = 2.26915;
            float beta = 1.0/temperature;
            TIMERSTART(AVERAGES)
            for(int j=0; j<RUNS_AVERAGE; j++)
            {   
                TIMERSTART(calculating)
                
                hipMemset(Observables, 0, sizeof(float)*2);              CUERR
                for(int i=0; i<OVERALL_ITERATIONS; i++)
                {
                    isis<<<gridDims, BLOCKSIZE>>>(Spins, 
                                                  length, 
                                                  deviceStates, 
                                                  i, 
                                                  beta);                  CUERR
                    hipDeviceSynchronize();                              CUERR
                    
                }
                getObservables<<<gridDims, BLOCKSIZE>>>(Spins, 
                                                        length, 
                                                        Observables);     CUERR
                hipMemcpy(&observables[t*4 + run*N_TEMPS*4 +2], 
                           Observables, sizeof(float)*2,
                           hipMemcpyDeviceToHost);                       CUERR
                observables[t*4 + run*N_TEMPS*4] = (float) length;
                observables[t*4 + run*N_TEMPS*4 +1] = temperature;
                observables[t*4 + run*N_TEMPS*4 +2] /= length*length*2;
                observables[t*4 + run*N_TEMPS*4 +3] /= length*length;
                
                energy += observables[t*4 + run*N_TEMPS*4 + 2];
                magnet += std::fabs(observables[t*4 + run*N_TEMPS*4 + 3]);
        
                TIMERSTOP(calculating)
                // Run a few more times to store more data
                TIMERSTART(additional_data)
                hipMemset(Stats, 0, sizeof(float)*2*DATA_PER_RUN);       CUERR
                printf("-");
                for(int k=0; k<DATA_PER_RUN; k++)
                {
                    for(int f=0; f<RUNS_BEFORE_FLUSH; f++)
                    {
                        isis<<<gridDims, BLOCKSIZE>>>(Spins, 
                                                  length, 
                                                  deviceStates, 
                                                  f, 
                                                  beta);                  CUERR
                        hipDeviceSynchronize();                          CUERR
                    }
                    
                    getObservables<<<gridDims, BLOCKSIZE>>>(Spins, 
                                                            length, 
                                                            &Stats[k*2]); CUERR
                   
                    if(k%4 == 0) {printf("\r (>^.^)> %d     ", k); fflush(stdout);}
                    else if(k%5 == 0) {printf("\r (>°.°)> ----"); fflush(stdout);}
                    
                    ///////     Dump Image
                    if(DUMP_IMG > 0)
                    {
                        if(k%DUMP_IMG == 0 && k > 0)
                        {
                            TIMERSTART(write_states_to_disk)
                            for (uint32_t l = 0; l < DUMP_IMG; l++) {
                                int idx = k-DUMP_IMG + l;
                                std::string image_name = "imgs/movie2/ising_size_" 
                                    + std::to_string(LENGTH) +  "_" + std::to_string(idx) + ".bmp";
                                dump_bitmap(&all_spins[l*LENGTH*LENGTH], LENGTH, LENGTH, image_name);
                            }
                            TIMERSTOP(write_states_to_disk)
                        }
                        hipMemcpy(all_spins+length*length*(k%DUMP_IMG), 
                                   Spins, sizeof(float)*length*length,
                                   hipMemcpyDeviceToHost);                   CUERR
                   }            
                }  
                
                TIMERSTOP(additional_data)
                // Dump data to plot statistics
                if(DATA_PER_RUN > 0 && !MC) 
                {  
                    hipMemcpy(stats, 
                               Stats, sizeof(float)*2*DATA_PER_RUN,
                               hipMemcpyDeviceToHost);                   CUERR    
                    for(int k=0; k<DATA_PER_RUN; k++)
                    {
                        stats[k*2] /= length*length*2;
                        stats[1 + k*2] /= length*length;
                        fs << length << "\t" << temperature
                           << "\t" << stats[k*2] << "\t" 
                           << stats[1 + k*2] << "\n";
                    }
                }
                // Dump data to use with xmgrace
                if(DATA_PER_RUN > 0 && MC)
                {
                    hipMemcpy(stats, 
                               Stats, sizeof(float)*2*DATA_PER_RUN,
                               hipMemcpyDeviceToHost);                   CUERR
                    for(int k=0; k<DATA_PER_RUN; k++)
                    {
                        stats[k*2] /= length*length*2;
                        stats[1 + k*2] /= length*length;
                        int mc = (k+1)*RUNS_BEFORE_FLUSH;
                        fs << mc << " " << stats[1 + k*2] << "\n";
                    }
                     
                }
                       
                // Reset spins
                hipMemcpy(Spins, spins, sizeof(float)*length*length, 
                   hipMemcpyHostToDevice);                               CUERR   
            }
            TIMERSTOP(AVERAGES)
            observables[t*4 + run*N_TEMPS*4 + 2] = energy/RUNS_AVERAGE;
            observables[t*4 + run*N_TEMPS*4 + 3] = magnet/RUNS_AVERAGE;
            printf("Lattice %d, temperature %f, energy %f, magnetization %f\n", 
                   length, temperature, observables[t*4 + run*N_TEMPS*4 +2], 
                   observables[t*4 + run*N_TEMPS*4 +3]);
        }
        
    }
    fs.close();

    std::ofstream myfile;
    if(argc > 1)
        myfile.open(argv[1]);
    else
        myfile.open("cuda_model");
    myfile << "lattice_size\ttemperature\tenergy\tmagnetization\n";
    for(int i=0; i<RUNS; i++)
    {
        for(int j=0; j<N_TEMPS; j++)
        {
            myfile << observables[j*4 + i*N_TEMPS*4] << "\t" 
                   << observables[j*4 + i*N_TEMPS*4 + 1] 
                   << "\t" << observables[j*4 + i*N_TEMPS*4 + 2] << "\t" 
                   << observables[j*4 + i*N_TEMPS*4 + 3] << "\n";
        }
        myfile << "Next\n";
    }
    myfile.close();

    hipHostFree(spins);                                                  CUERR
    hipHostFree(observables);                                            CUERR
    hipHostFree(all_spins);                                              CUERR
    hipFree(Spins);                                                      CUERR
    hipFree(deviceStates);                                               CUERR
    hipFree(Observables);                                                CUERR
}
